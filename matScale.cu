
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// scale kernel
__global__ void matScale(float *d_A, float *d_B, float scale, int N, int M) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// scale matrix elements
	if (row < N && col < M) {
		d_B[row * M + col] = d_A[row * M + col] / scale;
	}
}

int main() {
  	// var declaration
	int N = 3;
	int M = 3;
	float scale = 2.0f;
	float *A, *B;
	float *d_A, *d_B;

	// allocate host memory
	A = (float *)malloc(N * M * sizeof(float));
	B = (float *)malloc(N * M * sizeof(float));

	// allocate device memory
	hipMalloc(&d_A, N * M * sizeof(float));
	hipMalloc(&d_B, N * M * sizeof(float));

	// initialize data
	for (int i = 0; i < N * M; ++i) {
		A[i] = i - 3;
		B[i] = i;
	}
	
	// copy host data to device
	hipMemcpy(d_A, A, N * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N * M * sizeof(float), hipMemcpyHostToDevice);
        
	// launch kernel instance
	dim3 blockDim(16, 16);
	dim3 gridDim((M + blockDim.x - 1)/blockDim.x, (N + blockDim.y - 1)/blockDim.y);
	matScale<<<gridDim, blockDim>>>(d_A, d_B, scale, N, M);
	
	// copy result back to host
	hipMemcpy(A, d_A, N * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, N * M * sizeof(float), hipMemcpyDeviceToHost);

  	// display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("A: %f ", A[i * M + j]);
		}
		printf("\n");
	}

	printf("----------\n");
	printf("Matrix B: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("B: %f ", B[i * M + j]);
		}
		printf("\n");
	}
	
	// clean up data
	free(A); free(B);
	hipFree(d_A); hipFree(d_B);

	return 0;
}
