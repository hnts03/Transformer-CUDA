#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;

        if (prop.major > 3 || (prop.major == 3 && prop.minor >= 5)) {
            std::cout << "  Supports Dynamic Parallelism: Yes" << std::endl;
        } else {
            std::cout << "  Supports Dynamic Parallelism: No" << std::endl;
        }
    }

    return 0;
}

