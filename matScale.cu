#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// scale kernel
__global__ void matScale(float *d_A, float *d_B, float scale, int M, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// scale matrix elements
	if (row < M && col < N) {
		d_B[row * N + col] = d_A[row * N + col] / scale;
	}
}

int main() {
  	// var declaration
	int M = 3;
	int N = 3;
	float scale = 2.0f;
	float *h_A, *h_B;
	float *d_A, *d_B;

	// allocate host memory
	h_A = (float *)malloc(M * N * sizeof(float));
	h_B = (float *)malloc(M * N * sizeof(float));

	// allocate device memory
	hipMalloc(&d_A, M * N * sizeof(float));
	hipMalloc(&d_B, M * N * sizeof(float));

	// initialize data
	for (int i = 0; i < M * N; ++i) {
		h_A[i] = i - 3;
		h_B[i] = i;
	}
	
	// copy host data to device
	hipMemcpy(d_A, h_A, M * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, M * N * sizeof(float), hipMemcpyHostToDevice);
        
	// launch kernel instance
	dim3 blockDim(16, 16);
	dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (M + blockDim.y - 1)/blockDim.y);
	matScale<<<gridDim, blockDim>>>(d_A, d_B, scale, M, N);
	
	// copy result back to host
	hipMemcpy(A, d_A, M * N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, M * N * sizeof(float), hipMemcpyDeviceToHost);

  	// display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("A: %f ", A[i * M + j]);
		}
		printf("\n");
	}

	printf("----------\n");
	printf("Matrix B: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("B: %f ", B[i * M + j]);
		}
		printf("\n");
	}
	
	// clean up data
	free(A); free(B);
	hipFree(d_A); hipFree(d_B);

	return 0;
}
