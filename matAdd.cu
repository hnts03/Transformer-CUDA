#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// matrix addition kernel
__global__ void matAdd(float *d_A, float *d_B, float *d_C, int N, int M) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// add matrix elements
	if (row < N && col < M) {
		d_C[row * M + col] = d_A[row * M + col] + d_B[row * M + col];
	}
}

int main() {
  	// var declaration
	int N = 5;
	int M = 5;
	float *A, *B, *C;
	float *d_A, *d_B, *d_C;

	// allocate host memory
	A = (float *)malloc(N * M * sizeof(float));
	B = (float *)malloc(N * M * sizeof(float));
        C = (float *)malloc(N * M * sizeof(float));

	// allocate device memory
	hipMalloc(&d_A, N * M * sizeof(float));
	hipMalloc(&d_B, N * M * sizeof(float));
	hipMalloc(&d_C, N * M * sizeof(float));

	// initialize data
	for (int i = 0; i < N * M; ++i) {
		A[i] = i - 3;
		B[i] = i;
	}
	
	// copy host data to device
	hipMemcpy(d_A, A, N * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, N * M * sizeof(float), hipMemcpyHostToDevice);

	// launch kernel instance
	dim3 blockDim(16, 16);
	dim3 gridDim((M + blockDim.x - 1)/blockDim.x, (N + blockDim.y - 1)/blockDim.y);
	matAdd<<<gridDim, blockDim>>>(d_A, d_B, d_C, N, M);
	
	// copy result back to host
	hipMemcpy(A, d_A, N * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, N * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, N * M * sizeof(float), hipMemcpyDeviceToHost);

  	// display results
	printf("Matrix A: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("A: %f ", A[i * M + j]);
		}
		printf("\n)
	}

	printf("----------\n");
	printf("Matrix B: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("B: %f ", B[i * M + j]);
		}
		printf("\n)
	}

	printf("----------\n");
	printf("Matrix C: \n");
	printf("----------\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < M; ++j) {
			printf("C: %f ", C[i * M + j]);
		}
		printf("\n)
	}

	// clean up data
	free(A); free(B); free(C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);

	return 0;
}
