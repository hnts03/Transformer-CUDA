
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void softmax(float *d_in, float *d_out, float *expArr, float *redArr, int N) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < N) {
		float local_exp = expf(d_in[col]);
		expArr[col] = local_exp;
		redArr[col] = expArr[col];
		
		// amount of padding for parallel reduction
	        int padding = 0;
        	for (int e = 0; (float)N/(float)(1 << e)>= 1; ++e) {
                	padding = e + 1;
        	}

        	for (int i = 0; i < (1 << padding) - N; ++i) {
                	expArr[N + i] = 0;
			redArr[N + i] = 0;
        	}


		__syncthreads();	
		
		// parallel reduction to compute sum
		for (int stride = 1 << padding; stride >= 1; stride /= 2) {
			if (col < stride) {
				redArr[col] += redArr[col + stride];
			}
		}
		__syncthreads();
	}

	// calculate e^x / sum(e^x) = softmax for each element
	if (col < N) {
		float sum = redArr[0];
		d_out[col] = expArr[col] / sum;
	}
}

int main() {
	// var declaration
	int N = 6;
	float h_in[N];
	float h_out[N];
	float *d_in, *d_out;
	float *expArr;
	float *redArr;

	// memory allocation
	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));
	hipMalloc((void**)&expArr, 2 * N * sizeof(float));
	hipMalloc((void**)&redArr, 2 * N * sizeof(float));

	// data initialization
	for (int i = 0; i < N; ++i) {
		h_in[i] = (float)(rand() % 5 + 1);
	}
	
	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
	
	// launch softmax kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	softmax<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, expArr, redArr, N);

	// copy result to host
	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

	// print result
    	printf("Softmax input:\n");
    	for (int i = 0; i < N; ++i) {
        	printf("%f ", h_in[i]);
    	}
    	printf("\n----------\n");
  	printf("Softmax output:\n");
    	for (int i = 0; i < N; ++i) {
        	printf("%f ", h_out[i]);
    	}

    	printf("\n----------\n");

   	// compare with CPU implementation
    	float sum = 0.0f;
    	for (int i = 0; i < N; ++i) {
       	 	sum += exp(h_in[i]);
    	}
    	printf("Expected output:\n");
    	for (int i = 0; i < N; ++i) {
        	printf("%f ", exp(h_in[i]) / sum);
   	}
    	printf("\n");

	// clean device memory
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
